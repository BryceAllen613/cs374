/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 * Edited by Bryce Allen for proj09, cs374, Calvin University.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

 #include <stdio.h>

 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>
 // for timing
 #include <omp.h>
 
 /**
  * CUDA Kernel Device code
  *
  * Computes the vector addition of A and B into C. 
  * The 3 vectors have the same number of elements numElements.
  */
 __global__
 void vectorAdd(const float *A, const float *B, float *C, unsigned long numElements)
 {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
 }
 
 void checkErr(hipError_t err, const char* msg) 
 {
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
 }
 
 /**
  * Host main routine
  */
 int main(int argc, char** argv)
 {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    double startTime, endTime, totalTime = 0.0;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
    numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %lu elements]\n", numElements);
    printf("\n");

    // Allocate the host input vectors A & B
    float * h_A = (float *)malloc(size);
    float * h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");
    float * d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    checkErr(err, "Failed to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    //printf("Copy input data from the host memory to the CUDA device\n");

    startTime = omp_get_wtime();
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector B from host to device");
    endTime = omp_get_wtime() - startTime;

    printf("Copy time was %f\n", endTime);
    totalTime += endTime;
    startTime = 0.0;
    endTime = 0.0;

    // 3. Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    startTime = omp_get_wtime();
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    endTime = omp_get_wtime() - startTime;
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorAdd kernel");

    printf("Addition time was %f\n", endTime);
    totalTime += endTime;
    startTime = 0.0;
    endTime = 0.0;

    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    startTime = omp_get_wtime();
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    endTime = omp_get_wtime() - startTime;
    checkErr(err, "Failed to copy vector C from device to host");

    printf("Copy-back time was %f\n", endTime);
    totalTime += endTime;
    startTime = 0.0;
    endTime = 0.0;
    printf("total computation time was %f\n", totalTime);
    printf("\n");

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("CUDA test PASSED\n");
    //printf("CUDA time: %lf\n", stopTime-startTime); 

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Failed to free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");

    // repeat the computation sequentially
    startTime = omp_get_wtime();
    for (int i = 0; i < numElements; ++i)
    {
        h_C[i] = h_A[i] + h_B[i];
    }
    endTime = omp_get_wtime() - startTime;
    printf("total sequential computation time was %f\n", endTime);

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("\nNormal test PASSED\n");
//    printf("Normal time: %lf\n", stopTime-startTime); 
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
 }
 
 